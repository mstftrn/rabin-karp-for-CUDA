#include <stdio.h>
//#include <omp.h>
#include <string.h>
#include <math.h>
//#include "../common/common.h"
#include <hip/hip_runtime.h>

/*
 * compute string value, length should be small than strlen
 */
int compute_value(char *str, int length, int d, int q)
{
	int i = 0;
	int p0 = 0;

	for (i = 0; i < length; ++i) {
		p0 = (d * p0 + (str[i] /*- '0'*/ )) % q;
	}

	return p0;

}

int rk_matcher(char *str, char *pattern, int d, int q)
{
	int i = 0, j = 0;
	int str_length = strlen(str);
	int pattern_length = strlen(pattern);
	int p0 = 0;
	int ts[str_length];

	/* This code block prints what is inside the matrix
	   for (i=0;i<num_cores;i++)
	   {
	   for (j=0;j<el_chunk_len;j++)
	   if (tss[i][j]==0)
	   printf("%c", '0');
	   else
	   printf("%c", tss[i][j]);
	   printf("\n");
	   }
	 */

	//hash value of the pattern
	p0 = compute_value(pattern, pattern_length, d, q);

	//hash value of the first char
	ts[0] = compute_value(str, pattern_length, d, q);

	//p does not change, calculate once
	int p = pow(d, pattern_length - 1);
	for (i = 1; i < str_length - pattern_length + 1; i++) {
		ts[i] = ((str[i + pattern_length - 1]) * p
			 + (ts[i - 1] - (str[i - 1])) / d) % q;
		/*      (ts[i - 1] * d -
		   ((str[i - 1] - '0') * (int) pow(d,
		   pattern_length))) % q +
		   (str[i + pattern_length - 1]
		   - '0') % q; */
	}

/*    for (i=0;i<str_length-pattern_length+1;i++)
    {
    	printf("%d ", ts[i]);
    }*/

	for (i = 0; i <= str_length - pattern_length + 1; ++i) {
		if (ts[i] == p0) {
			for (j = 0; j < pattern_length; ++j) {
				if (pattern[j] != str[i + j]) {
					break;
				} else if (j == pattern_length - 1) {
					printf("%d\n", i);
				}
			}
		}
	}

	return 0;

}

__global__ void findHashes(char *d_css, int d_len, int *d_iss,
			   int pattern_length, int d, /*int q,*/ int p)
{
	int i = 0;
	int ind = d_len * threadIdx.x;
	d_iss += ind;
	d_css += ind;
	d_iss[0] = 0;
//      printf("%d %d %d %d %d %d", d_iss[0], d_len, pattern_length, d, q, p);
	int pw = 1;
	for (; i < pattern_length; i++) {
		d_iss[0] += pw * (d_css[i]);
		pw *= d;
	}
	//d_iss[0] %= q;
	//printf("%d ", d_iss[0]);

	for (i = 1; i < d_len - pattern_length + 1; i++) {
		d_iss[i] = ((d_css[i + pattern_length - 1]) * p
			    + (d_iss[i - 1] - (d_css[i - 1])) / d); //% q;
        //printf("%d ",d_iss[i]);
	}

}

__global__ void seekPattern(char *d_css, int d_len, int *d_iss,
                int pattern_length, char* pattern, int d, int p0) 
{
	int i = 0;
        int j=0;
	int ind = d_len * threadIdx.x;
	d_iss += ind;
	d_css += ind;

	for (i = 0; i < d_len - pattern_length + 1; i++) {
		if (d_iss[i] == p0) {
			for (j = 0; j < pattern_length; j++) {
				if (pattern[j] != d_css[i + j]) {
					break;
				} else if (j == pattern_length - 1) {

			//		printf("ThreadId: %d\n", threadIdx.x);
					printf("pos:%d\n", threadIdx.x*(d_len-pattern_length+1)+i-pattern_length+1);
				}
			}
		}
	}

}
int main(int argc, char *argv[])
{
	int i = 0;
	int j = 0;
	char str[] = "bababanaparaverbababanaparaverbababanaparaverbababanaparaverbababanaparaverbababanaparaverbababanaparaver";
	char pattern[] = "aba";
	int d = 3;
	//int q = 50000;
	int num_cores = 8;

	//CHECK(cudaDeviceReset());

	int str_length = strlen(str);
	//int nElem=str_length;
	int pattern_length = strlen(pattern);
	int chunk_len = (int)ceil((float)str_length / num_cores);
	int padding_len = chunk_len * num_cores - str_length;
	int el_chunk_len = chunk_len + pattern_length - 1;

	//matrix on host which holds the characters, each row will go to a core
	char css[num_cores][el_chunk_len];
	int iss[num_cores][el_chunk_len];
	//on the device
	char *d_css;
        char *d_pattern;
	//hashes on the device
	int *d_iss;
	int nchars = num_cores * el_chunk_len;
	hipMalloc((char **)&d_css, nchars * sizeof(char));
	hipMalloc((int **)&d_iss, nchars * sizeof(int));
        hipMalloc((char **)&d_pattern, pattern_length*sizeof(char));

	//initial zeroes
	for (i = 0; i < pattern_length - 1; i++)
		css[0][i] = 0;

	//first n-1 cores' characters
	for (i = 0; i < num_cores - 1; i++)
		for (j = 0; j < chunk_len; j++)
			css[i][j + pattern_length - 1] = str[i * chunk_len + j];

	//last core's characters
	for (i = (num_cores - 1) * chunk_len, j = 0; i < str_length; i++, j++)
		css[num_cores - 1][j + pattern_length - 1] = str[i];

	//last n-1 cores' padding characters
	for (i = 1; i < num_cores; i++)
		for (j = 0; j < pattern_length - 1; j++)
			css[i][j] = css[i - 1][j + chunk_len];

	//last core's last paddings
	for (i = 0; i < padding_len; i++)
		css[num_cores - 1][el_chunk_len - i - 1] = 0;

	//transfer css to device
	hipMemcpy(d_css, css, nchars, hipMemcpyHostToDevice);
	hipMemcpy(d_css, css, nchars, hipMemcpyHostToDevice);
	hipMemcpy(d_pattern, pattern, pattern_length, hipMemcpyHostToDevice);

	dim3 block(num_cores);	//str_length/pattern_length
	//__global__ void findHashes(char *d_css, int d_len, int *d_iss, int pattern_length, int d, int q, int p)
	int p = pow(d, pattern_length - 1);
	findHashes <<< 1, num_cores >>> (d_css, el_chunk_len, d_iss,
					 pattern_length, d, /*q,*/ p);

        //find the hash of the pattern
        int pw = 1;
        int p0=0;
        for (i=0; i < pattern_length; i++) {
            p0 += pw * (pattern[i]);
            pw *= d;
        }
	//printf("%d\n", p0);
        
        seekPattern<<<1, num_cores>>>(d_css, el_chunk_len, d_iss,
                pattern_length, d_pattern, d, p0); 

	//printf("%d %d %d %d %d \n", el_chunk_len, pattern_length, d, q, p);

	//cudaMemcpy(iss, d_iss, nchars * sizeof(int), cudaMemcpyDeviceToHost);
	/*for (i=0;i<num_cores;i++)
	   {
	   for (j=0;j<el_chunk_len;j++)
	   	printf("%d ", iss[i][j]);
	   printf("\n");
	   } 
	*/
	hipFree(d_iss);
	hipFree(d_css);

	//int pos = rk_matcher(str, pattern, d, q);
	//printf("%d", pos);
	return 0;
}
