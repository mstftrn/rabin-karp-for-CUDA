#include "hip/hip_runtime.h"
#include <stdio.h>
//#include <omp.h>
#include <string.h>
#include <math.h>
#include "../common/common.h"
#include <hip/hip_runtime.h>

/*
 * compute string value, length should be small than strlen
 */
int compute_value(char *str, int length, int d, int q)
{
    int i = 0;
    int p0 = 0;

    for (i = 0; i < length; ++i) {
	p0 = (d * p0 + (str[i] /*- '0'*/)) % q;
    }

    return p0;

}

int rk_matcher(char *str, char *pattern, int d, int q)
{
    int i = 0,j=0;
    int str_length = strlen(str);
    int pattern_length = strlen(pattern);
    int p0 = 0;
    int ts[str_length];

    /* This code block prints what is inside the matrix
    for (i=0;i<num_cores;i++)
    {
        for (j=0;j<el_chunk_len;j++)
            if (tss[i][j]==0)
                printf("%c", '0');
            else
                printf("%c", tss[i][j]);
        printf("\n");
    }
    */



    //hash value of the pattern
    p0 = compute_value(pattern, pattern_length, d, q);
    
    //hash value of the first char
    ts[0] = compute_value(str, pattern_length, d, q);

    //p does not change, calculate once
    int p=pow(d, pattern_length-1);
    for (i = 1; i < str_length-pattern_length+1; i++) 
    {
	ts[i] = ((str[i + pattern_length - 1])*p
                    +(ts[i-1]-(str[i-1]))/d)%q;
	/*	(ts[i - 1] * d -
		 ((str[i - 1] - '0') * (int) pow(d,
						 pattern_length))) % q +
		(str[i + pattern_length - 1]
		 - '0') % q;*/
    }

/*    for (i=0;i<str_length-pattern_length+1;i++)
    {
    	printf("%d ", ts[i]);
    }*/

    for (i = 0; i <= str_length - pattern_length+1; ++i) {
	if (ts[i] == p0) {
	    for (j = 0; j < pattern_length; ++j) {
		if (pattern[j] != str[i + j]) {
		    break;
		} else if (j == pattern_length - 1) {
		    printf("%d\n", i);
		}
	    }
	}
    }

    return 0;

}

__global__ void findHashes(char *d_css, char *pattern, int d, int q)
{
    printf("Hello World from GPU!\n");
}

int main(int argc, char *argv[])
{
    int i=0;
    int j=0;
    char * str="bababanaparaver";
    char * pattern="aba";
    int prime=3;
    int q=50;
    int num_cores=4;

    findHashes<<<1, num_cores>>>(d_css, pattern, d, q);

    //CHECK(hipDeviceReset());

    int str_length = strlen(str);
    int nElem=str_length;
    int pattern_length = strlen(pattern);
    int chunk_len=(int)ceil((float)str_length/num_cores);
    int padding_len=chunk_len*num_cores-str_length;
    int el_chunk_len=chunk_len+pattern_length-1;

    //matrix on host which holds the characters, each row will go to a core
    char css[num_cores][el_chunk_len];
    //on the device
    char *d_css;
    //hashes on the device
    int *d_iss;
    int nchars=num_cores*el_chunk_len;
    hipMalloc((char **)&d_css, nchars*sizeof(char));
    hipMalloc((int **)&d_iss, nchars*sizeof(int));
    
    //initial zeroes
    for (i=0; i<pattern_length-1; i++)
    	css[0][i]=0;

    //first n-1 cores' characters
    for (i=0; i<num_cores-1; i++)
        for (j=0;j<chunk_len;j++)
            css[i][j+pattern_length-1]=str[i*chunk_len+j];
    
    //last core's characters
    for (i=(num_cores-1)*chunk_len, j=0; i<str_length;i++,j++)
        css[num_cores-1][j+pattern_length-1]=str[i];
    
    //last n-1 cores' padding characters
    for (i=1;i<num_cores;i++)
        for (j=0;j<pattern_length-1;j++)
            css[i][j]=css[i-1][j+chunk_len];
    
    //last core's last paddings
    for (i=0; i<padding_len;i++)
        css[num_cores-1][el_chunk_len-i-1]=0;

    //transfer css to device
    hipMemcpy(d_css, css, nchars, hipMemcpyHostToDevice);

    dim3 block (num_cores);//str_length/pattern_length

    int pos = rk_matcher(str, pattern, prime, q);
    //printf("%d", pos);
    return 0;
}

